//******************************************************************************
//
// File:  MetricCenterGpu.cu
// This file is a part of project 3 of the course:Foundation of Parallel Computing,
// under taken in Fall 2015 at Rochester Institute of Technology. 
//******************************************************************************


#include <hip/hip_runtime.h>
#include <float.h>

//Threads per block
#define ThreadsPerBlock 1024

//Structre for storing point with x,y cordinates
typedef struct{
	double x;
	double y;
	}
	point_t;
	
//Structure for storing reduced metric center point's radius and index	
typedef struct{
	double radius;
	int index;
	}reducedRadius_t;
	
//Find distance between two points; 
//square_root( (b1-a1)^2 + (b2-a2)^2 ).
//Returns distance between points a and b
__device__ double pointDistance
	(point_t *a, point_t *b){
		return sqrt( ( (b->x - a->x)*(b->x - a->x)+
			(b->y - a->y)*(b->y - a->y)));
			 
	}

//Find and save min radius and index;
//min( a->radius , b->radius ) or if no min, min( a->index , b->index )	
__device__ void saveMinReducedRadius
	(reducedRadius_t *a, reducedRadius_t *b){
		if( a->radius > b->radius ){
			a->radius = b->radius;
			a->index = b->index;
		}else if( a->radius == b->radius ){
			if( a->index > b->index ){
				a->index = b->index;
			}
		}
	}
//Find and save max radius and index;
//max(a->radius, b->radius)
__device__ void saveMaxReducedRadius
	(reducedRadius_t *a, reducedRadius_t *b){
		if( a->radius < b->radius ){
			a->radius = b->radius;
			a->index = b->index;
		}
	}

//Per thread shared variables	
__shared__ reducedRadius_t shrdMaxRadius [ThreadsPerBlock];



/**
 * kernel program for find metric center from given points
 * This kernel function is called with 1-D grid of size same
 * as number of multiprocessor and 1-D blocks with number of 
 * threads as ThreadsPerBlock.
 *
 * @param  points   Array of points.
 * @param  size     Length of points array.
 * @param  minOfMaxRadius     Array for saving reduced radius and Index.
 *
 * @author  Karan Bhagat
 * @version 6-Nov-2015
 */
extern "C" __global__ void findMetricCenter
	(point_t *points, int size, reducedRadius_t *minOfMaxRadius){

	int blockNum = blockIdx.x;
	int totalBlocks = gridDim.x;	
	int thrdIdx = threadIdx.x;  // Index of this thread in a block
	
	minOfMaxRadius[blockNum].radius = DBL_MAX;
	minOfMaxRadius[blockNum].index = 0; 
	
	reducedRadius_t currentMaxRadius = {0.0,0};

	//find max possible distance of points and then find point with minimum radius.	
	for( int i = blockNum ; i < size; i += totalBlocks ){
		
		shrdMaxRadius[thrdIdx].radius = 0.0;
		shrdMaxRadius[thrdIdx].index = 0;
		
		currentMaxRadius.index = i;
		
		for( int j = thrdIdx ; j < size; j += ThreadsPerBlock ){
			if( j == i ) continue;
			currentMaxRadius.radius = pointDistance( &points[i], &points[j]);
			saveMaxReducedRadius(&shrdMaxRadius[thrdIdx],&currentMaxRadius);
		}
		__syncthreads();
		
		//in block reduction for each point
		for( int i = ThreadsPerBlock/2; i > 0; i >>= 1){
			if( thrdIdx < i ){
				saveMaxReducedRadius(&shrdMaxRadius[thrdIdx],&shrdMaxRadius[thrdIdx+i]);
			
			}
			__syncthreads();
		}
		
		//saving min radius found in block
		if( thrdIdx == 0 ){
			saveMinReducedRadius(&minOfMaxRadius[blockNum], &shrdMaxRadius[thrdIdx]);
		}
	}	
}
	